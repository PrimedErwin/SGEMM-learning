#include "hip/hip_runtime.h"
﻿//This is a CUBLAS matrix multiplication program
//running on WINDOWS

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__


#include "hip/hip_runtime.h"
#include "hipblas.h"
#include ""

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <Windows.h>

//basic param
#define M 1024
#define N 2048
#define K 512
#define DIFF 1e-3

//CUDA error check
void checkerror(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "CUDA error at %d: ", __LINE__);
		fprintf(stderr, "%s, %s\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	return;
}

void randomInit(float* data, int size)
{
	for (int i = 0; i < size; i++) data[i] = rand() / (float)RAND_MAX;
}

struct matSize
{
	int wA, wB, wC, hA, hB, hC;
	matSize(int wwA = 0, int hhA = 0, int wwB = 0, int hhB = 0, int wwC = 0, int hhC = 0) :wA(wwA),
		hA(hhA), wB(wwB), hB(hhB), wC(wwC), hC(hhC) {}
};


void checkMulResult(float* hC, float* dC, int width, int height)
{
	int cnt = 0;
	for (int j = 0; j < height; j++)
	{
		for (int i = 0; i < width; i++)
		{
			int index = j * width + i;
			float diff = fabs(hC[index] - dC[index]);
			if (diff > DIFF)
			{
				cnt++;
				if(cnt<21) fprintf(stderr, "hC result = %f\tdC result = %f\n", hC[index], dC[index]);
			}
		}
	}
	fprintf(stderr, "Error result count = %d\n", cnt);
}


/// <summary>
/// Performs small matrix mul by CPU for result checking
/// </summary>
/// <param name="A">=matrix A</param>
/// <param name="B">=matrix B</param>
/// <param name="C">=matrix C</param>
/// <param name="hA">=M, height of A</param>
/// <param name="wB">=N, width of B</param>
/// <param name="wA">=K, width of A, height of B</param>
void matrixMulCPU(const float* A, const float* B, float* C, matSize &matsize)
{
	double sum;
	for (int k = 0; k < matsize.wA; k++)
	{
		for (int m = 0; m < matsize.hA; m++)
		{
			for (int n = 0; n < matsize.wB; n++)
			{
				sum = A[m * matsize.wA + k] * B[k * matsize.wB + n];
				C[m * matsize.wB + n] += (float)sum;
			}
		}
	}
}

#pragma comment (lib, "cublas.lib")
int main(int argc, char** argv)
{
	printf("Starting CUBLAS matrix multiplication on device 0: ");
	int devID = 0;
	hipSetDevice(devID);
	hipDeviceProp_t Prop;
	hipGetDeviceProperties(&Prop, devID);
	printf("\"%s\" compute capability %d.%d\n", Prop.name, Prop.major, Prop.minor);
	
	//basic things
	matSize matsize(K, M, N, K, M, N);
	printf("Matrix size: \n\
		%d x %d * %d x %d\n",
		matsize.hA, matsize.wA, matsize.hB, matsize.wB);
	srand(time(nullptr));
	unsigned int sizeA = matsize.hA * matsize.wA;
	unsigned int sizeB = matsize.hB * matsize.wB;
	unsigned int sizeC = matsize.hC * matsize.wC;
	unsigned int nByteA = sizeA * sizeof(float);
	unsigned int nByteB = sizeB * sizeof(float);
	unsigned int nByteC = sizeC * sizeof(float);
	static const float alpha = 1.0f;
	static const float beta = .0f;

	//basic stopwatch
	LARGE_INTEGER IpFreq, IpStart, IpEnd;
	hipEvent_t cuStart, cuEnd;
	hipEventCreate(&cuStart);
	hipEventCreate(&cuEnd);
	float cpu_time, gpu_time;
	QueryPerformanceFrequency(&IpFreq);

	//define matrix
	float* hA = (float*)malloc(nByteA);
	float* hB = (float*)malloc(nByteB);
	float* hC = (float*)malloc(nByteC);
	float* hC_from_gpu = (float*)malloc(nByteC);
	float* dA, * dB, * dC;
	hipMallocAsync(&dA, nByteA, hipStreamPerThread);
	hipMallocAsync(&dB, nByteB, hipStreamPerThread);
	hipMallocAsync(&dC, nByteC, hipStreamPerThread);
	checkerror("Malloc Error");

	//init matrix
	randomInit(hA, sizeA);
	randomInit(hB, sizeB);
	memset(hC, 0, nByteC);
	hipMemcpyAsync(dA, hA, nByteA, hipMemcpyHostToDevice, hipStreamPerThread);
	hipMemcpyAsync(dB, hB, nByteB, hipMemcpyHostToDevice, hipStreamPerThread);

	//calculate by CPU
	QueryPerformanceCounter(&IpStart);
	matrixMulCPU(hA, hB, hC, matsize);
	QueryPerformanceCounter(&IpEnd);
	cpu_time = (double)(IpEnd.QuadPart - IpStart.QuadPart) * 1e3 / IpFreq.QuadPart;
	printf("CPU time cost %.4f ms\n\n", cpu_time);
	hipDeviceSynchronize();
	checkerror("Sync error");

	//CUBLAS setup
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);
	checkerror("CUBLAS create error");

	//calculate by GPU, warmup first
	hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		matsize.wB, matsize.hA, matsize.wA,
		&alpha, dB, matsize.wB,
		dA, matsize.wA, &beta,
		dC, matsize.wB);
	checkerror("CUBLAS warmup failed");

	//calculate by GPU
	hipEventRecord(cuStart);
	for (int i = 0; i < 50; i++)
	{
		hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			matsize.wB, matsize.hA, matsize.wA,
			&alpha, dB, matsize.wB,
			dA, matsize.wA, &beta,
			dC, matsize.wB);
	}
	hipEventRecord(cuEnd);
	hipEventSynchronize(cuEnd);
	hipEventElapsedTime(&gpu_time, cuStart, cuEnd);
	checkerror("Event sync error");
	gpu_time /= 50;
	printf("GPU time cost %.4f ms\n\n", gpu_time);

	//compare result
	hipMemcpy(hC_from_gpu, dC, nByteC, hipMemcpyDeviceToHost);
	checkMulResult(hC, hC_from_gpu, matsize.wC, matsize.hC);
	double flops = (double)matsize.hA * (double)matsize.wA * (double)matsize.wB * 2.0;
	double gigaFlops = (flops * 1.0e-9) / (gpu_time / 1000.0);
	printf("FP32 %.4f TFlop/s, Ops %.4f Ops\n", gigaFlops/1000.0, flops);


	//free matrix
	free(hA);
	free(hB);
	free(hC);
	hipFreeAsync(dA, hipStreamPerThread);
	hipFreeAsync(dB, hipStreamPerThread);
	hipFreeAsync(dC, hipStreamPerThread);
	hipblasDestroy(cublasHandle);
	hipEventDestroy(cuStart);
	hipEventDestroy(cuEnd);
	
	return 0;
}